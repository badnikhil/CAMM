#include <iostream>
#include <hip/hip_runtime.h>
#include "../Header/matmul_kernels.cuh"
#include <iomanip>
#include <stdexcept>

#define N 4096

void random_fill(float* arr, int n) {
    for (int i = 0; i < n; ++i) arr[i] = static_cast<float>(rand()) / RAND_MAX;
}

int main() {
    size_t bytes = N * N * sizeof(float);
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        std::cerr << "Host memory allocation failed!" << std::endl;
        if (h_A) free(h_A);
        if (h_B) free(h_B);
        if (h_C) free(h_C);
        return 1;
    }
    random_fill(h_A, N*N);
    random_fill(h_B, N*N);

    float *d_A, *d_B, *d_C;
    hipError_t err;
    err = hipMalloc(&d_A, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_A: " << hipGetErrorString(err) << std::endl;
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMalloc(&d_B, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMalloc(&d_C, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Memcpy H2D failed for A: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Memcpy H2D failed for B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }

    dim3 threads(16, 16);
    dim3 blocks((N+15)/16, (N+15)/16);

    // CUDA events for timing
    hipEvent_t start, stop, h2d_start, h2d_stop, d2h_start, d2h_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);

    // Host to Device timing
    hipEventRecord(h2d_start);
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipEventRecord(h2d_stop);
    hipEventSynchronize(h2d_stop);
    float h2d_ms = 0;
    hipEventElapsedTime(&h2d_ms, h2d_start, h2d_stop);

    // Warmup kernel run
    matmul_naive<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Kernel timing (average over 10 runs)
    float kernel_ms = 0;
    int runs = 1;
    for (int i = 0; i < runs; ++i) {
        hipEventRecord(start);
        matmul_naive<<<blocks, threads>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        kernel_ms += ms;
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
            break;
        }
    }
    kernel_ms /= runs;

    // Device to Host timing
    hipEventRecord(d2h_start);
    err = hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Memcpy D2H failed for C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    hipEventRecord(d2h_stop);
    hipEventSynchronize(d2h_stop);
    float d2h_ms = 0;
    hipEventElapsedTime(&d2h_ms, d2h_start, d2h_stop);

    std::cout << "C[0] = " << h_C[0] << std::endl;
    std::cout << "Host to Device memcpy time: " << h2d_ms << " ms" << std::endl;
    std::cout << "Kernel execution time (avg of " << runs << "): " << kernel_ms << " ms" << std::endl;
    std::cout << "Device to Host memcpy time: " << d2h_ms << " ms" << std::endl;

    // GFLOPS calculation
    double gflops = (2.0 * N * N * N) / (kernel_ms / 1000.0) / 1e9;
    std::cout << std::fixed << std::setprecision(8);
    std::cout << "Kernel GFLOPS: " << gflops << std::endl;

    // Checksum calculation
    double checksum = 0.0;
    for (int i = 0; i < N*N; ++i) checksum += h_C[i];
    std::cout << "Checksum: " << checksum << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(h2d_start);
    hipEventDestroy(h2d_stop);
    hipEventDestroy(d2h_start);
    hipEventDestroy(d2h_stop);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
} 