#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../Header/matmul_kernels.cuh"
#include <iomanip>
#include <stdexcept>

void random_fill(float* arr, int n) {
    for (int i = 0; i < n; ++i) arr[i] = static_cast<float>(rand()) / RAND_MAX;
}


void checksum(const float *A, float *B, float *C, int n) {
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < n; ++k) {
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
    double checksum = 0.0;
    for (int i = 0; i < n*n; ++i) checksum += C[i];
    std::cout << "Checksum CPU: " << checksum << std::endl;
}

int run_benchmark(int N) {
    size_t bytes = N * N * sizeof(float);
    float *h_A = (float*)malloc(bytes);
    float *h_B = (float*)malloc(bytes);
    float *h_C = (float*)malloc(bytes);
    if (!h_A || !h_B || !h_C) {
        std::cerr << "Host memory allocation failed!" << std::endl;
        if (h_A) free(h_A);
        if (h_B) free(h_B);
        if (h_C) free(h_C);
        return 1;
    }
    random_fill(h_A, N*N);
    random_fill(h_B, N*N);
    

    checksum(h_A , h_B , h_C , N);
    
    // when accessing B matrix the memory around it is also copied..
    //  this will increase cache hits because for multiplying a(0.0)
    //  full row 0 of B will be faster loaded
    float *d_A, *d_B, *d_C;
    hipError_t err;
    err = hipMalloc(&d_A, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_A: " << hipGetErrorString(err) << std::endl;
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMalloc(&d_B, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMalloc(&d_C, bytes);
    if (err != hipSuccess) {
        std::cerr << "Device memory allocation failed for d_C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Memcpy H2D failed for A: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    err = hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Memcpy H2D failed for B: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    int x = 16;
    int y = 16;
    int z = 1;
    dim3 threads(x, y, z);
    printf("dimensions of threads: %d, %d, %d\n", x, y, z);
    dim3 blocks(ceil(N/(float)(x)), ceil(N/(float)(y)) , 1);

    // CUDA events for timing
    hipEvent_t start, stop, h2d_start, h2d_stop, d2h_start, d2h_stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&h2d_start);
    hipEventCreate(&h2d_stop);
    hipEventCreate(&d2h_start);
    hipEventCreate(&d2h_stop);

    // Host to Device timing
    hipEventRecord(h2d_start);
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);
    hipEventRecord(h2d_stop);
    hipEventSynchronize(h2d_stop);
    float h2d_ms = 0;
    hipEventElapsedTime(&h2d_ms, h2d_start, h2d_stop);

    // Warmup kernel run
    matmul_coalesced<<<blocks, threads>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    // Kernel timing (average over 10 runs)
    float kernel_ms = 0;
    int runs = 10;
    for (int i = 0; i < runs; ++i) {
        hipEventRecord(start);
        matmul_coalesced<<<blocks, threads>>>(d_A, d_B, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        kernel_ms += ms;
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
            break;
        }
    }
    kernel_ms /= runs;

    // Device to Host timing
    hipEventRecord(d2h_start);
    err = hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Memcpy D2H failed for C: " << hipGetErrorString(err) << std::endl;
        hipFree(d_A); hipFree(d_B); hipFree(d_C);
        free(h_A); free(h_B); free(h_C);
        return 1;
    }
    hipEventRecord(d2h_stop);
    hipEventSynchronize(d2h_stop);
    float d2h_ms = 0;
    hipEventElapsedTime(&d2h_ms, d2h_start, d2h_stop);

    std::cout << "C[0] = " << h_C[0] << std::endl;
    std::cout << "Host to Device memcpy time: " << h2d_ms << " ms" << std::endl;
    std::cout << "Kernel execution time (avg of " << runs << "): " << kernel_ms << " ms" << std::endl;
    std::cout << "Device to Host memcpy time: " << d2h_ms << " ms" << std::endl;

    // GFLOPS calculation
    double gflops = (2.0 * N * N * N) / (kernel_ms / 1000.0) / 1e9;
    std::cout << std::fixed << std::setprecision(8);
    std::cout << "Kernel GFLOPS: " << gflops << std::endl;

    // Checksum calculation
    double checksum = 0.0;
    for (int i = 0; i < N*N; ++i) checksum += h_C[i];
    std::cout << "Checksum: " << checksum << std::endl<<std::endl << std::endl<<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(h2d_start);
    hipEventDestroy(h2d_stop);
    hipEventDestroy(d2h_start);
    hipEventDestroy(d2h_stop);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
} 


int main() {
    int sizes[] = {512, 1024, 2048, 3072, 4096, 6144, 8192};
    for (int N : sizes) {
        std::cout  << "Running benchmark for N = " << N << std::endl ;
        run_benchmark(N);
    }
    return 0;
}